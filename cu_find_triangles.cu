#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>

void load(char* file);
__global__ void count(int *A, int *colind,int *block_sums, int nnz);
void gen_colind();

int *A;
int nnz;
int *colind;

struct timeval startwtime, endwtime;
double seq_time;

#define TPB 1024
#define NB 1024

__global__ void count(int *A, int *colind, int *block_sums, int nnz){

    int idx = blockDim.x*blockIdx.x + threadIdx.x;
    extern __shared__ int cache[];

    int csum = 0;
    int i,j,k,l;
    int nextk,nextl;

    while(idx < nnz){
        i = A[idx];
        j = A[nnz + idx];

        k = colind[j-1];
        l = colind[i-1];

        nextk = (j == A[2*nnz-1])?nnz:colind[j];
        nextl = (i == A[2*nnz-1])?nnz:colind[i];

        do{
            if(A[k] > A[l]){
                l++;
            }
            else if(A[k] < A[l]){
                k++;
            }
            else{
                csum++;
                k++;
                l++;
            }
        }while(k<nextk && l<nextl);

        idx += blockDim.x*gridDim.x;
    }

    cache[threadIdx.x] = csum;
    __syncthreads();

    //per-block Reduction
    for(int s = blockDim.x/2;s>0;s>>=1){

        if(threadIdx.x < s){
            cache[threadIdx.x] += cache[threadIdx.x + s];
            __syncthreads();
        }

    }
    if(threadIdx.x == 0) block_sums[blockIdx.x] = cache[0];
}

int main(int argc, char **argv){

    if(argc != 2){
        printf("Usage: %s [filename]\n",argv[0]);
        printf("Quiting...\n");
        exit(1);
    }

    load(argv[1]);
    gen_colind();

    int nthreads_block = TPB;
    int nblocks = NB;
    int nt;
    int *block_sums = (int *)malloc(nblocks*sizeof(int));

    int *d_A,*d_colind,*d_block_sums;

    hipMalloc((void **)&d_A,2*nnz*sizeof(int));
    hipMalloc((void **)&d_colind,A[2*nnz-1]*sizeof(int));
    hipMalloc((void **)&d_block_sums,nblocks*sizeof(int));

    hipMemcpy(d_A,A,2*nnz*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(d_colind,colind,A[2*nnz-1]*sizeof(int),hipMemcpyHostToDevice);

    //Call kernel and measure time passed
    gettimeofday (&startwtime, NULL);
    count<<<nblocks,nthreads_block,nthreads_block*sizeof(int)>>>(d_A,d_colind,d_block_sums,nnz);
    hipDeviceSynchronize();
    gettimeofday (&endwtime, NULL);

    //Copy partial sums to host
    hipMemcpy(block_sums, d_block_sums, nblocks*sizeof(int), hipMemcpyDeviceToHost);

    //Sum all block sums to solve problem
    nt = 0;
    for(int i=0;i<nblocks;i++){
        nt += block_sums[i];
    }
    nt /= 6;

    seq_time = (double)((endwtime.tv_usec - startwtime.tv_usec)/1.0e3
				+ (endwtime.tv_sec - startwtime.tv_sec)*1.0e3);

    printf("Found %d triangles in %f ms\n",nt,seq_time);

    free(A);
    free(colind);
    free(block_sums);
    hipFree(d_A);
    hipFree(d_colind);
    hipFree(d_block_sums);
}

void load(char* file){
    FILE *fp;
    int size;

    if((fp = fopen(file,"rb")) == NULL){
        printf("Failed to open file.\nExiting...\n");
        exit(1);
    }

    fseek(fp,0,SEEK_END);
    size = ftell(fp);
    size /= 4;
    fseek(fp,0,SEEK_SET);
    A = (int *)malloc(size*sizeof(int));

    int i = 0;
    int nread;

    for(i=0;i<size;i++){
        nread = fread(&A[i],sizeof(int),1,fp);
        if(nread != 1){
            printf("Error reading file!\nExiting...\n");
            printf("%d\n",i);
            exit(1);
        }
    }

    nnz = size/2;
    fclose(fp);
}

void gen_colind(){

    int lastcol = A[2*nnz-1];
    colind = (int *)malloc(lastcol*sizeof(int));

    int prev = 0;
    for(int i=0;i<nnz;i++){
        if(A[nnz+i] != prev){
            prev = A[nnz+i];
            colind[prev-1] = i;
        }
    }

}
